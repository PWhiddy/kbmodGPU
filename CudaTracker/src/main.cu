#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : KBMOD CUDA
 Author      : Peter Whidden
 Description :
 ============================================================================
 */

#include <iostream>
#include <fstream>
#include <numeric>
#include <stdlib.h>
#include <cstdlib>
#include <sstream>
#include <ctime>
#include <math.h>
//#include <algorithm>

#include <fitsio.h>
#include "GeneratorPSF.h"
#include "FakeAsteroid.h"

void writeFitsImg(fitsfile *f, const char *name, long fpix, 
	long *naxes, long nelements, void *array);

const char* parseLine(std::ifstream& cFile, int debug);

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/*
 * A struct to represent a potential trajectory
 */
struct trajectory {
	// Trajectory velocities
	float xVel; 
	float yVel;
	// Likelyhood
	float lh;
	// Origin
	int x; 
	int y;
	// Number of images summed
	int itCount; 
};

/* 
 * For comparing trajectory structs, so that they can be sorted
 */
int compareTrajectory( const void * a, const void * b)
{
        return (int)(5000.0*(((trajectory*)b)->lh - ((trajectory*)a)->lh));
}

/*
 * Device kernel that compares the provided PSF distribution to the distribution
 * around each pixel in the provided image
 */
__global__ void convolvePSF(int width, int height, int imageCount,
	float *image, float *results, float *psf, int psfRad, 
	int psfDim, float background, float normalization)
{
	// Find bounds of convolution area
	const int x = blockIdx.x*32+threadIdx.x;
	const int y = blockIdx.y*32+threadIdx.y;
	const int minX = max(x-psfRad, 0);
	const int minY = max(y-psfRad, 0);
	const int maxX = min(x+psfRad, width-1);
	const int maxY = min(y+psfRad, height-1);
	const int dx = maxX-minX;
	const int dy = maxY-minY;
	if (dx < 1 || dy < 1 ) return;
 
	// Read kernel
	float sumDifference = 0.0;
	for (int j=minY; j<=maxY; ++j)
	{
		// #pragma unroll
		for (int i=minX; i<=maxX; ++i)
		{
			sumDifference += (image[j*width+i] - background)
					 * psf[(j-minY)*psfDim+i-minX];
		}
	}

	results[y*width+x] = sumDifference*normalization;

}

/*
 * Searches through images (represented as a flat array of floats) looking for most likely
 * trajectories in the given list. Outputs a results image of best trajectories. Note that
 * for now only the single best trajectory starting at each pixel makes it to results. 
 */
__global__ void searchImages(int width, int height, int imageCount, float *images, 
	int trajectoryCount, trajectory *tests, trajectory *results, int edgePadding)
{

	// Get trajectory origin
	int x = blockIdx.x*32+threadIdx.x;
	int y = blockIdx.y*32+threadIdx.y;
	// Give up if any trajectories will hit image edges
	if (x < edgePadding || x + edgePadding > width ||
	    y < edgePadding || y + edgePadding > height) return;

	trajectory best = { .xVel = 0.0, .yVel = 0.0, .lh = 0.0, 
			     .x = x, .y = y, .itCount = trajectoryCount };
	
	for (int t=0; t<trajectoryCount; ++t)
	{
		float xVel = tests[t].xVel;
		float yVel = tests[t].yVel;
		float currentLikelyhood = 0.0;
		for (int i=0; i<imageCount; ++i)
		{
			currentLikelyhood += logf( images[ i*width*height + 
				(y+int( yVel*float(i)))*width +
				 x + int( xVel*float(i)) ] ); 	
		}
		
		if ( currentLikelyhood > best.lh )
		{
			best.lh = currentLikelyhood;
			best.xVel = xVel;
			best.yVel = yVel;
		}		
	}	
	
	results[ y*width + x ] = best;	

}


int main(int argc, char* argv[])
{

	/* Read parameters from config file */
	std::ifstream pFile ("parameters.config");
    	if (!pFile.is_open()) 
		std::cout << "Unable to open parameters file." << '\n';
	
	int debug             = atoi(parseLine(pFile, false));
	int imageCount        = atoi(parseLine(pFile, debug));
	float psfSigma        = atof(parseLine(pFile, debug));
	float asteroidLevel   = atof(parseLine(pFile, debug));
	float initialX        = atof(parseLine(pFile, debug));
	float initialY        = atof(parseLine(pFile, debug));
	float velocityX       = atof(parseLine(pFile, debug));
	float velocityY       = atof(parseLine(pFile, debug));
	float backgroundLevel = atof(parseLine(pFile, debug));
	float backgroundSigma = atof(parseLine(pFile, debug));
	int anglesCount       = atoi(parseLine(pFile, debug));
	int velocitySteps     = atoi(parseLine(pFile, debug));
	float minVelocity     = atof(parseLine(pFile, debug));
	float maxVelocity     = atof(parseLine(pFile, debug));
	int writeFiles        = atoi(parseLine(pFile, debug));
	std::string origPath  = parseLine(pFile, debug);
	std::string psiPath   = parseLine(pFile, debug);
	pFile.close();
     
	/* Create instances of psf and object generators */
	GeneratorPSF *gen = new GeneratorPSF();

	psfMatrix testPSF = gen->createGaussian(psfSigma);

	float psfCoverage = gen->printPSF(testPSF, debug);

	FakeAsteroid *asteroid = new FakeAsteroid();


	/* Setup Image/FITS Properties of test Images  */
	fitsfile *fptr;
	long fpixel = 1, /*naxis = 2,*/ nelements;//, exposure;
	long naxes[2] = { 1024, 1024 }; // X and Y dimensions
	nelements = naxes[0] * naxes[1];
	std::stringstream ss;
	float **pixelArray = new float*[imageCount];

	// Create asteroid images //
	for (int imageIndex=0; imageIndex<imageCount; ++imageIndex)
	{

		/* Initialize the values in the image with noisy astro */

		pixelArray[imageIndex] = new float[nelements];
		asteroid->createImage( pixelArray[imageIndex], naxes[0], naxes[1],
	 	    	velocityX*float(imageIndex)+initialX,  // Asteroid X position 
			velocityY*float(imageIndex)+initialY, // Asteroid Y position
			testPSF, asteroidLevel, backgroundLevel, backgroundSigma);

	}

	/*
	// Load real image into first slot
	ss << "../realImg.fits";
	fits_open_data(&fptr, ss.str().c_str(), READONLY, &status);
	fits_report_error(stderr, status);
	ss.str("");
	ss.clear();
	*/


	/* Generate psi images on device */

	std::clock_t t1 = std::clock();

	// Pointers to device memory //
	float **result = new float*[nelements];
	float *devicePsf;
	float *deviceImageSource;
	float *deviceImageResult;

	dim3 blocks(32,32);
	dim3 threads(32,32);

	// Allocate Device memory
	CUDA_CHECK_RETURN(hipMalloc((void **)&devicePsf, sizeof(float)*testPSF.dim*testPSF.dim));
	CUDA_CHECK_RETURN(hipMalloc((void **)&deviceImageSource, sizeof(float)*nelements));
	CUDA_CHECK_RETURN(hipMalloc((void **)&deviceImageResult, sizeof(float)*nelements));

	CUDA_CHECK_RETURN(hipMemcpy(devicePsf, testPSF.kernel,
		sizeof(float)*testPSF.dim*testPSF.dim, hipMemcpyHostToDevice));

	for (int procIndex=0; procIndex<imageCount; ++procIndex)
	{

		result[procIndex] = new float[nelements];
		// Copy image to
		CUDA_CHECK_RETURN(hipMemcpy(deviceImageSource, pixelArray[procIndex],
			sizeof(float)*nelements, hipMemcpyHostToDevice));

		convolvePSF<<<blocks, threads>>> (naxes[0], naxes[1], 
			imageCount, deviceImageSource, deviceImageResult, devicePsf, 
			testPSF.dim/2, testPSF.dim, backgroundLevel, 1.0/psfCoverage);

		CUDA_CHECK_RETURN(hipMemcpy(result[procIndex], deviceImageResult,
			sizeof(float)*nelements, hipMemcpyDeviceToHost));
	}

	CUDA_CHECK_RETURN(hipFree(devicePsf));
	CUDA_CHECK_RETURN(hipFree(deviceImageSource));
	CUDA_CHECK_RETURN(hipFree(deviceImageResult));

	std::clock_t t2 = std::clock();

	std::cout << imageCount << " images, " <<
		1000.0*(t2 - t1)/(double) (CLOCKS_PER_SEC*imageCount) 
		  << " ms per image\n";


	///* Search images on GPU *///
	
	std::clock_t t3 = std::clock();
		
	/* Create test trajectories */
	float *angles = new float[anglesCount];
	for (int i=0; i<anglesCount; ++i)
	{
		angles[i] = 6.283185*float(i)/float(anglesCount);
	}

	float *velocities = new float[velocitySteps];
	float dv = (maxVelocity-minVelocity)/float(velocitySteps);
	for (int i=0; i<velocitySteps; ++i)
	{
		velocities[i] = minVelocity+float(i)*dv;	
	}	
 
	int trajCount = anglesCount*velocitySteps;
	trajectory *trajTests = new trajectory[trajCount];
	for (int a=0; a<anglesCount; ++a)
	{
		for (int v=0; v<velocitySteps; ++v)
		{
			trajTests[a*velocitySteps+v].xVel = cos(angles[a])*velocities[v];
			trajTests[a*velocitySteps+v].yVel = sin(angles[a])*velocities[v]; 
		}
	}

	// Allocate Host memory to store results in
	trajectory* trajResult = new trajectory[nelements];

	// Allocate Device memory 
	trajectory *deviceTests;
	trajectory *deviceSearchResults;
	float *deviceImages;

	CUDA_CHECK_RETURN(hipMalloc((void **)&deviceTests, sizeof(trajectory)*trajCount));
	CUDA_CHECK_RETURN(hipMalloc((void **)&deviceImages, sizeof(float)*nelements*imageCount));
	CUDA_CHECK_RETURN(hipMalloc((void **)&deviceSearchResults, sizeof(trajectory)*nelements));
	

	// Copy trajectories to search
	CUDA_CHECK_RETURN(hipMemcpy(deviceTests, trajTests,
			sizeof(trajectory)*trajCount, hipMemcpyHostToDevice));

	// Copy over psi images one at a time
	for (int i=0; i<imageCount; ++i)
	{
		CUDA_CHECK_RETURN(hipMemcpy(deviceImages+nelements*i, result[i],
			sizeof(float)*nelements, hipMemcpyHostToDevice));
	}

	// assumes object is not moving more than 2 pixels per image
	int padding = 2*imageCount+int(psfSigma)+1;

	// Launch Search
	searchImages<<<blocks, threads>>> (naxes[0], naxes[1], imageCount, deviceImages,
				trajCount, deviceTests, deviceSearchResults, padding);

	// Read back results
	CUDA_CHECK_RETURN(hipMemcpy(trajResult, deviceSearchResults,
				sizeof(trajectory)*nelements, hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipFree(deviceTests));
	CUDA_CHECK_RETURN(hipFree(deviceSearchResults));
	CUDA_CHECK_RETURN(hipFree(deviceImages));

	
	// Sort results by likelihood
	qsort(trajResult, nelements, sizeof(trajectory), compareTrajectory);
	if (debug)
	{
		for (int i=0; i<15; ++i)
		{
			if (i+1 < 10) std::cout << " ";
			std::cout << i+1 << ". Likelihood: "  << trajResult[i].lh 
				  << " at x: " << trajResult[i].x << ", y: " << trajResult[i].y
				  << "  and velocity x: " << trajResult[i].xVel 
				  << ", y: " << trajResult[i].yVel << "\n" ;
		}
	}

	std::clock_t t4 = std::clock();

	std::cout << imageCount << " images, " <<
		1.0*(t4 - t3)/(double) (CLOCKS_PER_SEC) << " seconds to test " 
		<< trajCount << " possible trajectories starting from " 
		<< ((naxes[0]-padding)*(naxes[1]-padding)) << " pixels. " << "\n";

	std::cout << "Writing images to file... ";

	// Write images to file 
	if (writeFiles)
	{
		for (int writeIndex=0; writeIndex<imageCount; ++writeIndex)
		{
			/* Create file name */
			ss << origPath << "T";
			// Add leading zeros to filename
			if (writeIndex+1<100) ss << "0";
			if (writeIndex+1<10) ss << "0";
			ss << writeIndex+1 << ".fits";
			writeFitsImg(fptr, ss.str().c_str(), fpixel, naxes, 
				nelements, pixelArray[writeIndex]);
			ss.str("");
			ss.clear();		

			ss << psiPath << "T";
			if (writeIndex+1<100) ss << "0";
			if (writeIndex+1<10) ss << "0"; 
			ss << writeIndex+1 << "psi.fits";
			writeFitsImg(fptr, ss.str().c_str(), fpixel, naxes, 
				nelements, pixelArray[writeIndex]);
			ss.str("");
			ss.clear();
		}
	}
	std::cout << "Done.\n";

	/* Write results file */
	// std::cout needs to be rerouted to output to console after this...
	std::freopen("results.txt", "w", stdout);
	std::cout << "# t0_x t0_y theta_par theta_perp v_x v_y likelihood est_flux\n";
        for (int i=0; i<20; ++i)
        {
                std::cout << trajResult[i].x << " " << trajResult[i].y << " 0.0 0.0 "
                          << trajResult[i].xVel << " " << trajResult[i].yVel << " "       
                          << trajResult[i].lh << " 0.0\n" ;
        }

	// Finished!

	/* Free memory */
	for (int im=0; im<imageCount; ++im)
	{
		delete[] pixelArray[im];
		delete[] result[im];
	}

	delete[] pixelArray;
	delete[] result;
	
	delete[] angles;
	delete[] velocities;
	delete[] trajTests;	
	delete[] trajResult;
	
	delete gen;
	delete asteroid;

	return 0;
} 

const char* parseLine(std::ifstream& pFile, int debug)
{
	std::string line;
	getline(pFile, line);
        int delimiterPos = line.find(":");
	if (debug) 
	{
		std::cout << line.substr(0, delimiterPos );
		std::cout << " : " << line.substr(delimiterPos + 2) << "\n";
	}
	return (line.substr(delimiterPos + 2)).c_str();
}

void writeFitsImg(fitsfile *f, const char *name, long fpix, long *naxes, long nelements, void *array)
{
	/* initialize status before calling fitsio routines */
	int status = 0;
        /* Create file with name */
	fits_create_file(&f, name, &status);

	/* Create the primary array image (32-bit float pixels */
	fits_create_img(f, FLOAT_IMG, 2 /*naxis*/, naxes, &status);

	/* Write the array of floats to the image */
	fits_write_img(f, TFLOAT, 1, nelements, array, &status);
	fits_close_file(f, &status);
	fits_report_error(stderr, status);
}


/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err)
			<< "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

